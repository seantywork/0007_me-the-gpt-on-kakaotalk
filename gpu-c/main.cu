// CUDA 12.1

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
// #include <helper_functions.h>
//#include <helper_cuda.h>
/*
__global__
void saxpy(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

*/
__global__
void insanely_long_division(int N, int *out, int *x, int *y)
{


    for(int i=0;i<N;i++){

        printf("[%d, %d]: ",\
            blockIdx.y*gridDim.x+blockIdx.x,\
            threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x);

        int count = 0;
        int top = x[i];
        int bottom = y[i];

        for (int j=bottom;j<top;j++){

            if (top % j == 0){
                count += 1;
            } else {
                continue;
            }

        }

        out[i] = count;

        printf("%d\n",count);
    }


}




int main(int argc, char **argv)
{

    int devID;
    hipDeviceProp_t props;

    // This will pick the best possible CUDA capable device
//    devID = findCudaDevice(argc, (const char **)argv);

    //Get GPU information
//    checkCudaErrors(cudaGetDevice(&devID));
//    checkCudaErrors(cudaGetDeviceProperties(&props, devID));
    hipGetDevice(&devID);
    hipGetDeviceProperties(&props, devID);
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",
            devID, props.name, props.major, props.minor);

    printf("printf() is called. Output:\n\n");

    int N = 100;

    int A = 0;

    int *res, *x, *y, *d_res, *d_x, *d_y;

    res = (int*)malloc(N*sizeof(int));
    x = (int*)malloc(N*sizeof(int));
    y = (int*)malloc(N*sizeof(int));

    hipMalloc(&d_res, N*sizeof(int)); 
    hipMalloc(&d_x, N*sizeof(int)); 
    hipMalloc(&d_y, N*sizeof(int));

    for (int i = 0; i < N; i++) {
        res[i] = 0;
        x[i] = 100;
        y[i] = 5;
    }

    hipMemcpy(d_res, res, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(int), hipMemcpyHostToDevice);


    //  saxpy<<<(N+255)/256, 256>>>(N, 2.0f, d_x, d_y);

    printf("counting total insane division..........\n");

    insanely_long_division<<<N, 1>>>(N, d_res, d_x, d_y);

    hipDeviceSynchronize();

    hipMemcpy(res, d_res, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(x, d_x, N*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(y, d_y, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        A += res[i];
    
    }

    printf("\ntotal insane division count: %d\n", A);
    printf("\ncompleted!\n");

    hipFree(d_res);
    hipFree(d_x);
    hipFree(d_y);
    free(res);
    free(x);
    free(y);



    return EXIT_SUCCESS;
}
